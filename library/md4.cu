#include "hip/hip_runtime.h"
#include <cudassl/md4.h>

#if defined(CUDASSL_MD4_C)

__device__ void md4_init(md4_context *ctx)
{
    memset(ctx, 0, sizeof(md4_context));
}

__device__ void md4_free(md4_context *ctx)
{
    if (ctx == NULL)
        return;

    zeroize(ctx, sizeof(md4_context));
}

/*
 * MD4 context setup
 */
__device__ void md4_starts(md4_context *ctx)
{
    ctx->total[0] = 0;
    ctx->total[1] = 0;

    ctx->state[0] = 0x67452301;
    ctx->state[1] = 0xEFCDAB89;
    ctx->state[2] = 0x98BADCFE;
    ctx->state[3] = 0x10325476;
}

__device__ void md4_process(md4_context *ctx, const unsigned char data[64])
{
    uint32_t X[16], A, B, C, D;

    GET_UINT32_LE(X[ 0], data,  0);
    GET_UINT32_LE(X[ 1], data,  4);
    GET_UINT32_LE(X[ 2], data,  8);
    GET_UINT32_LE(X[ 3], data, 12);
    GET_UINT32_LE(X[ 4], data, 16);
    GET_UINT32_LE(X[ 5], data, 20);
    GET_UINT32_LE(X[ 6], data, 24);
    GET_UINT32_LE(X[ 7], data, 28);
    GET_UINT32_LE(X[ 8], data, 32);
    GET_UINT32_LE(X[ 9], data, 36);
    GET_UINT32_LE(X[10], data, 40);
    GET_UINT32_LE(X[11], data, 44);
    GET_UINT32_LE(X[12], data, 48);
    GET_UINT32_LE(X[13], data, 52);
    GET_UINT32_LE(X[14], data, 56);
    GET_UINT32_LE(X[15], data, 60);

#define S(x,n) ((x << n) | ((x & 0xFFFFFFFF) >> (32 - n)))

    A = ctx->state[0];
    B = ctx->state[1];
    C = ctx->state[2];
    D = ctx->state[3];

#define F(x, y, z) ((x & y) | ((~x) & z))
#define P(a,b,c,d,x,s) { a += F(b,c,d) + x; a = S(a,s); }

    P(A, B, C, D, X[ 0],  3);
    P(D, A, B, C, X[ 1],  7);
    P(C, D, A, B, X[ 2], 11);
    P(B, C, D, A, X[ 3], 19);
    P(A, B, C, D, X[ 4],  3);
    P(D, A, B, C, X[ 5],  7);
    P(C, D, A, B, X[ 6], 11);
    P(B, C, D, A, X[ 7], 19);
    P(A, B, C, D, X[ 8],  3);
    P(D, A, B, C, X[ 9],  7);
    P(C, D, A, B, X[10], 11);
    P(B, C, D, A, X[11], 19);
    P(A, B, C, D, X[12],  3);
    P(D, A, B, C, X[13],  7);
    P(C, D, A, B, X[14], 11);
    P(B, C, D, A, X[15], 19);

#undef P
#undef F

#define F(x,y,z) ((x & y) | (x & z) | (y & z))
#define P(a,b,c,d,x,s) { a += F(b,c,d) + x + 0x5A827999; a = S(a,s); }

    P(A, B, C, D, X[ 0],  3);
    P(D, A, B, C, X[ 4],  5);
    P(C, D, A, B, X[ 8],  9);
    P(B, C, D, A, X[12], 13);
    P(A, B, C, D, X[ 1],  3);
    P(D, A, B, C, X[ 5],  5);
    P(C, D, A, B, X[ 9],  9);
    P(B, C, D, A, X[13], 13);
    P(A, B, C, D, X[ 2],  3);
    P(D, A, B, C, X[ 6],  5);
    P(C, D, A, B, X[10],  9);
    P(B, C, D, A, X[14], 13);
    P(A, B, C, D, X[ 3],  3);
    P(D, A, B, C, X[ 7],  5);
    P(C, D, A, B, X[11],  9);
    P(B, C, D, A, X[15], 13);

#undef P
#undef F

#define F(x,y,z) (x ^ y ^ z)
#define P(a,b,c,d,x,s) { a += F(b,c,d) + x + 0x6ED9EBA1; a = S(a,s); }

    P(A, B, C, D, X[ 0],  3);
    P(D, A, B, C, X[ 8],  9);
    P(C, D, A, B, X[ 4], 11);
    P(B, C, D, A, X[12], 15);
    P(A, B, C, D, X[ 2],  3);
    P(D, A, B, C, X[10],  9);
    P(C, D, A, B, X[ 6], 11);
    P(B, C, D, A, X[14], 15);
    P(A, B, C, D, X[ 1],  3);
    P(D, A, B, C, X[ 9],  9);
    P(C, D, A, B, X[ 5], 11);
    P(B, C, D, A, X[13], 15);
    P(A, B, C, D, X[ 3],  3);
    P(D, A, B, C, X[11],  9);
    P(C, D, A, B, X[ 7], 11);
    P(B, C, D, A, X[15], 15);

#undef F
#undef P

    ctx->state[0] += A;
    ctx->state[1] += B;
    ctx->state[2] += C;
    ctx->state[3] += D;
}

/*
 * MD4 process buffer
 */
__device__ void md4_update(md4_context *ctx, const unsigned char *input, size_t ilen)
{
    size_t fill;
    uint32_t left;

    if (ilen == 0)
        return;

    left = ctx->total[0] & 0x3F;
    fill = 64 - left;

    ctx->total[0] += (uint32_t) ilen;
    ctx->total[0] &= 0xFFFFFFFF;

    if (ctx->total[0] < (uint32_t) ilen)
        ctx->total[1]++;

    if (left && ilen >= fill)
    {
        memcpy((void *) (ctx->buffer + left),
                (void *) input, fill);
        md4_process(ctx, ctx->buffer);
        input += fill;
        ilen  -= fill;
        left = 0;
    }

    while(ilen >= 64)
    {
        md4_process(ctx, input);
        input += 64;
        ilen  -= 64;
    }

    if (ilen > 0)
    {
        memcpy((void *) (ctx->buffer + left),
                (void *) input, ilen);
    }
}

__constant__ static const unsigned char md4_padding[64] =
{
 0x80, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0
};

/*
 * MD4 final digest
 */
__device__ void md4_finish(md4_context *ctx, unsigned char output[16])
{
    uint32_t last, padn;
    uint32_t high, low;
    unsigned char msglen[8];

    high = (ctx->total[0] >> 29)
         | (ctx->total[1] <<  3);
    low  = (ctx->total[0] <<  3);

    PUT_UINT32_LE(low,  msglen, 0);
    PUT_UINT32_LE(high, msglen, 4);

    last = ctx->total[0] & 0x3F;
    padn = (last < 56) ? (56 - last) : (120 - last);

    md4_update(ctx, (unsigned char *) md4_padding, padn);
    md4_update(ctx, msglen, 8);

    PUT_UINT32_LE(ctx->state[0], output,  0);
    PUT_UINT32_LE(ctx->state[1], output,  4);
    PUT_UINT32_LE(ctx->state[2], output,  8);
    PUT_UINT32_LE(ctx->state[3], output, 12);
}

/*
 * output = MD4(input buffer)
 */
__device__ void md4(const unsigned char *input, size_t ilen, unsigned char output[16])
{
    md4_context ctx;

    md4_init(&ctx);
    md4_starts(&ctx);
    md4_update(&ctx, input, ilen);
    md4_finish(&ctx, output);
    md4_free(&ctx);
}

#if defined(CUDASSL_SELF_TEST)

/*
 * RFC 1320 test vectors
 */
__constant__ static unsigned char md4_test_buf[7][81] =
{
    { "" },
    { "a" },
    { "abc" },
    { "message digest" },
    { "abcdefghijklmnopqrstuvwxyz" },
    { "ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz0123456789" },
    { "12345678901234567890123456789012345678901234567890123456789012" \
      "345678901234567890" }
};

__constant__ static const int md4_test_buflen[7] =
{
    0, 1, 3, 14, 26, 62, 80
};

static const unsigned char md4_test_sum[7][16] =
{
    { 0x31, 0xD6, 0xCF, 0xE0, 0xD1, 0x6A, 0xE9, 0x31,
      0xB7, 0x3C, 0x59, 0xD7, 0xE0, 0xC0, 0x89, 0xC0 },
    { 0xBD, 0xE5, 0x2C, 0xB3, 0x1D, 0xE3, 0x3E, 0x46,
      0x24, 0x5E, 0x05, 0xFB, 0xDB, 0xD6, 0xFB, 0x24 },
    { 0xA4, 0x48, 0x01, 0x7A, 0xAF, 0x21, 0xD8, 0x52,
      0x5F, 0xC1, 0x0A, 0xE8, 0x7A, 0xA6, 0x72, 0x9D },
    { 0xD9, 0x13, 0x0A, 0x81, 0x64, 0x54, 0x9F, 0xE8,
      0x18, 0x87, 0x48, 0x06, 0xE1, 0xC7, 0x01, 0x4B },
    { 0xD7, 0x9E, 0x1C, 0x30, 0x8A, 0xA5, 0xBB, 0xCD,
      0xEE, 0xA8, 0xED, 0x63, 0xDF, 0x41, 0x2D, 0xA9 },
    { 0x04, 0x3F, 0x85, 0x82, 0xF2, 0x41, 0xDB, 0x35,
      0x1C, 0xE6, 0x27, 0xE1, 0x53, 0xE7, 0xF0, 0xE4 },
    { 0xE3, 0x3B, 0x4D, 0xDC, 0x9C, 0x38, 0xF2, 0x19,
      0x9C, 0x3E, 0x7B, 0x16, 0x4F, 0xCC, 0x05, 0x36 }
};

/*
 * Checkup routine
 */
__global__ void md4_self_test_kernel(unsigned char *outputs)
{
    int i;
    unsigned char *md4sum;

    for (i = 0; i < 6; i++) {
        md4sum = outputs + (TX * 6 + i) * MD4_DIGEST_LENGTH;

        md4(md4_test_buf[i], md4_test_buflen[i], md4sum);
    }
}

int md4_self_test(int verbose, cuda_device *d) {
  int i; 
  unsigned char ret[2][3][MD4_DIGEST_LENGTH];

  md4_self_test_kernel<<<1, 2>>>(d->device_data_out);
  cuda_download_data(ret, d->device_data_out, 2 * 3 * MD4_DIGEST_LENGTH);

  for (i = 0; i < 6; i++) {
    if (verbose != 0)
      printf("  MD4 test #%d: ", i + 1);

    if (memcmp(ret[0][i], md4_test_sum[i], MD4_DIGEST_LENGTH) != 0 &&
      memcmp(ret[1][i], md4_test_sum[i], MD4_DIGEST_LENGTH) != 0) {
      if (verbose != 0)
        printf("failed\n");
    } else {
      if (verbose != 0)
        printf("passed\n");
    }
  }

  if (verbose != 0)
    printf("\n");

  return 0;
}

#define DATASIZE 1000L
#define LOOPS 100000L
__global__ void md4_performance_test_kernel() {
  int i;
  unsigned char src[8];
  unsigned char ret[MD4_DIGEST_LENGTH];
  md4_context ctx;


  md4_init(&ctx);
  md4_starts(&ctx);

  memset(src, 0, 8);

  for (i = 0; i < LOOPS; i++)
    md4_update(&ctx, src, 8);

  md4_finish(&ctx, ret);
  md4_free(&ctx);
}

extern "C" int md4_performance_test(int verbose, cuda_device *d) {
  int i; float h;

  CUDA_START_TIME

  // for (i = 0; i < LOOPS; ++i) {
  md4_performance_test_kernel<<<DATASIZE, MAX_THREAD>>>();
  // }

  CUDA_STOP_TIME("  MD4")
  printf("    Block Data size: %ld\n", MAX_THREAD * 8 * DATASIZE);
  printf("    Block Loops: %ld\n", LOOPS);

  TALK_LIKE_A_HUMAN_BEING(MAX_THREAD * 8 * DATASIZE * LOOPS, "    ", " in total\n");
  TALK_LIKE_A_HUMAN_BEING(MAX_THREAD * 8 * DATASIZE * LOOPS / gpu_time * 1000, "    ", "/sec\n");
  
  printf("    %ld loops in total\n", LOOPS * MAX_THREAD * DATASIZE);
  printf("    %f loops/sec\n", LOOPS * MAX_THREAD * DATASIZE / gpu_time * 1000);

  if (verbose != 0)
    printf("\n");

  return 0;
}

#endif /* CUDASSL_SELF_TEST */

#endif /* CUDASSL_MD4_C */

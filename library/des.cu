#include "hip/hip_runtime.h"
#include "cudassl/des.h"

#if defined(CUDASSL_DES_C)

/*
 * Expanded DES S-boxes
 */
__constant__ static const uint32_t SB1[64] = {
  0x01010400, 0x00000000, 0x00010000, 0x01010404,
  0x01010004, 0x00010404, 0x00000004, 0x00010000,
  0x00000400, 0x01010400, 0x01010404, 0x00000400,
  0x01000404, 0x01010004, 0x01000000, 0x00000004,
  0x00000404, 0x01000400, 0x01000400, 0x00010400,
  0x00010400, 0x01010000, 0x01010000, 0x01000404,
  0x00010004, 0x01000004, 0x01000004, 0x00010004,
  0x00000000, 0x00000404, 0x00010404, 0x01000000,
  0x00010000, 0x01010404, 0x00000004, 0x01010000,
  0x01010400, 0x01000000, 0x01000000, 0x00000400,
  0x01010004, 0x00010000, 0x00010400, 0x01000004,
  0x00000400, 0x00000004, 0x01000404, 0x00010404,
  0x01010404, 0x00010004, 0x01010000, 0x01000404,
  0x01000004, 0x00000404, 0x00010404, 0x01010400,
  0x00000404, 0x01000400, 0x01000400, 0x00000000,
  0x00010004, 0x00010400, 0x00000000, 0x01010004
};

__constant__ static const uint32_t SB2[64] = {
  0x80108020, 0x80008000, 0x00008000, 0x00108020,
  0x00100000, 0x00000020, 0x80100020, 0x80008020,
  0x80000020, 0x80108020, 0x80108000, 0x80000000,
  0x80008000, 0x00100000, 0x00000020, 0x80100020,
  0x00108000, 0x00100020, 0x80008020, 0x00000000,
  0x80000000, 0x00008000, 0x00108020, 0x80100000,
  0x00100020, 0x80000020, 0x00000000, 0x00108000,
  0x00008020, 0x80108000, 0x80100000, 0x00008020,
  0x00000000, 0x00108020, 0x80100020, 0x00100000,
  0x80008020, 0x80100000, 0x80108000, 0x00008000,
  0x80100000, 0x80008000, 0x00000020, 0x80108020,
  0x00108020, 0x00000020, 0x00008000, 0x80000000,
  0x00008020, 0x80108000, 0x00100000, 0x80000020,
  0x00100020, 0x80008020, 0x80000020, 0x00100020,
  0x00108000, 0x00000000, 0x80008000, 0x00008020,
  0x80000000, 0x80100020, 0x80108020, 0x00108000
};

__constant__ static const uint32_t SB3[64] = {
  0x00000208, 0x08020200, 0x00000000, 0x08020008,
  0x08000200, 0x00000000, 0x00020208, 0x08000200,
  0x00020008, 0x08000008, 0x08000008, 0x00020000,
  0x08020208, 0x00020008, 0x08020000, 0x00000208,
  0x08000000, 0x00000008, 0x08020200, 0x00000200,
  0x00020200, 0x08020000, 0x08020008, 0x00020208,
  0x08000208, 0x00020200, 0x00020000, 0x08000208,
  0x00000008, 0x08020208, 0x00000200, 0x08000000,
  0x08020200, 0x08000000, 0x00020008, 0x00000208,
  0x00020000, 0x08020200, 0x08000200, 0x00000000,
  0x00000200, 0x00020008, 0x08020208, 0x08000200,
  0x08000008, 0x00000200, 0x00000000, 0x08020008,
  0x08000208, 0x00020000, 0x08000000, 0x08020208,
  0x00000008, 0x00020208, 0x00020200, 0x08000008,
  0x08020000, 0x08000208, 0x00000208, 0x08020000,
  0x00020208, 0x00000008, 0x08020008, 0x00020200
};

__constant__ static const uint32_t SB4[64] = {
  0x00802001, 0x00002081, 0x00002081, 0x00000080,
  0x00802080, 0x00800081, 0x00800001, 0x00002001,
  0x00000000, 0x00802000, 0x00802000, 0x00802081,
  0x00000081, 0x00000000, 0x00800080, 0x00800001,
  0x00000001, 0x00002000, 0x00800000, 0x00802001,
  0x00000080, 0x00800000, 0x00002001, 0x00002080,
  0x00800081, 0x00000001, 0x00002080, 0x00800080,
  0x00002000, 0x00802080, 0x00802081, 0x00000081,
  0x00800080, 0x00800001, 0x00802000, 0x00802081,
  0x00000081, 0x00000000, 0x00000000, 0x00802000,
  0x00002080, 0x00800080, 0x00800081, 0x00000001,
  0x00802001, 0x00002081, 0x00002081, 0x00000080,
  0x00802081, 0x00000081, 0x00000001, 0x00002000,
  0x00800001, 0x00002001, 0x00802080, 0x00800081,
  0x00002001, 0x00002080, 0x00800000, 0x00802001,
  0x00000080, 0x00800000, 0x00002000, 0x00802080
};

__constant__ static const uint32_t SB5[64] = {
  0x00000100, 0x02080100, 0x02080000, 0x42000100,
  0x00080000, 0x00000100, 0x40000000, 0x02080000,
  0x40080100, 0x00080000, 0x02000100, 0x40080100,
  0x42000100, 0x42080000, 0x00080100, 0x40000000,
  0x02000000, 0x40080000, 0x40080000, 0x00000000,
  0x40000100, 0x42080100, 0x42080100, 0x02000100,
  0x42080000, 0x40000100, 0x00000000, 0x42000000,
  0x02080100, 0x02000000, 0x42000000, 0x00080100,
  0x00080000, 0x42000100, 0x00000100, 0x02000000,
  0x40000000, 0x02080000, 0x42000100, 0x40080100,
  0x02000100, 0x40000000, 0x42080000, 0x02080100,
  0x40080100, 0x00000100, 0x02000000, 0x42080000,
  0x42080100, 0x00080100, 0x42000000, 0x42080100,
  0x02080000, 0x00000000, 0x40080000, 0x42000000,
  0x00080100, 0x02000100, 0x40000100, 0x00080000,
  0x00000000, 0x40080000, 0x02080100, 0x40000100
};

__constant__ static const uint32_t SB6[64] = {
  0x20000010, 0x20400000, 0x00004000, 0x20404010,
  0x20400000, 0x00000010, 0x20404010, 0x00400000,
  0x20004000, 0x00404010, 0x00400000, 0x20000010,
  0x00400010, 0x20004000, 0x20000000, 0x00004010,
  0x00000000, 0x00400010, 0x20004010, 0x00004000,
  0x00404000, 0x20004010, 0x00000010, 0x20400010,
  0x20400010, 0x00000000, 0x00404010, 0x20404000,
  0x00004010, 0x00404000, 0x20404000, 0x20000000,
  0x20004000, 0x00000010, 0x20400010, 0x00404000,
  0x20404010, 0x00400000, 0x00004010, 0x20000010,
  0x00400000, 0x20004000, 0x20000000, 0x00004010,
  0x20000010, 0x20404010, 0x00404000, 0x20400000,
  0x00404010, 0x20404000, 0x00000000, 0x20400010,
  0x00000010, 0x00004000, 0x20400000, 0x00404010,
  0x00004000, 0x00400010, 0x20004010, 0x00000000,
  0x20404000, 0x20000000, 0x00400010, 0x20004010
};

__constant__ static const uint32_t SB7[64] = {
  0x00200000, 0x04200002, 0x04000802, 0x00000000,
  0x00000800, 0x04000802, 0x00200802, 0x04200800,
  0x04200802, 0x00200000, 0x00000000, 0x04000002,
  0x00000002, 0x04000000, 0x04200002, 0x00000802,
  0x04000800, 0x00200802, 0x00200002, 0x04000800,
  0x04000002, 0x04200000, 0x04200800, 0x00200002,
  0x04200000, 0x00000800, 0x00000802, 0x04200802,
  0x00200800, 0x00000002, 0x04000000, 0x00200800,
  0x04000000, 0x00200800, 0x00200000, 0x04000802,
  0x04000802, 0x04200002, 0x04200002, 0x00000002,
  0x00200002, 0x04000000, 0x04000800, 0x00200000,
  0x04200800, 0x00000802, 0x00200802, 0x04200800,
  0x00000802, 0x04000002, 0x04200802, 0x04200000,
  0x00200800, 0x00000000, 0x00000002, 0x04200802,
  0x00000000, 0x00200802, 0x04200000, 0x00000800,
  0x04000002, 0x04000800, 0x00000800, 0x00200002
};

__constant__ static const uint32_t SB8[64] = {
  0x10001040, 0x00001000, 0x00040000, 0x10041040,
  0x10000000, 0x10001040, 0x00000040, 0x10000000,
  0x00040040, 0x10040000, 0x10041040, 0x00041000,
  0x10041000, 0x00041040, 0x00001000, 0x00000040,
  0x10040000, 0x10000040, 0x10001000, 0x00001040,
  0x00041000, 0x00040040, 0x10040040, 0x10041000,
  0x00001040, 0x00000000, 0x00000000, 0x10040040,
  0x10000040, 0x10001000, 0x00041040, 0x00040000,
  0x00041040, 0x00040000, 0x10041000, 0x00001000,
  0x00000040, 0x10040040, 0x00001000, 0x00041040,
  0x10001000, 0x00000040, 0x10000040, 0x10040000,
  0x10040040, 0x10000000, 0x00040000, 0x10001040,
  0x00000000, 0x10041040, 0x00040040, 0x10000040,
  0x10040000, 0x10001000, 0x10001040, 0x00000000,
  0x10041040, 0x00041000, 0x00041000, 0x00001040,
  0x00001040, 0x00040040, 0x10000000, 0x10041000
};

/*
 * PC1: left and right halves bit-swap
 */
static const uint32_t LHs[16] = {
  0x00000000, 0x00000001, 0x00000100, 0x00000101,
  0x00010000, 0x00010001, 0x00010100, 0x00010101,
  0x01000000, 0x01000001, 0x01000100, 0x01000101,
  0x01010000, 0x01010001, 0x01010100, 0x01010101
};

static const uint32_t RHs[16] = {
  0x00000000, 0x01000000, 0x00010000, 0x01010000,
  0x00000100, 0x01000100, 0x00010100, 0x01010100,
  0x00000001, 0x01000001, 0x00010001, 0x01010001,
  0x00000101, 0x01000101, 0x00010101, 0x01010101,
};

/*
 * Initial Permutation macro
 */
#define DES_IP(X,Y) {                                         \
  T = ((X >>  4) ^ Y) & 0x0F0F0F0F; Y ^= T; X ^= (T <<  4);   \
  T = ((X >> 16) ^ Y) & 0x0000FFFF; Y ^= T; X ^= (T << 16);   \
  T = ((Y >>  2) ^ X) & 0x33333333; X ^= T; Y ^= (T <<  2);   \
  T = ((Y >>  8) ^ X) & 0x00FF00FF; X ^= T; Y ^= (T <<  8);   \
  Y = ((Y << 1) | (Y >> 31)) & 0xFFFFFFFF;                    \
  T = (X ^ Y) & 0xAAAAAAAA; Y ^= T; X ^= T;                   \
  X = ((X << 1) | (X >> 31)) & 0xFFFFFFFF;                    \
}

/*
 * Final Permutation macro
 */
#define DES_FP(X,Y) {                                         \
  X = ((X << 31) | (X >> 1)) & 0xFFFFFFFF;                    \
  T = (X ^ Y) & 0xAAAAAAAA; X ^= T; Y ^= T;                   \
  Y = ((Y << 31) | (Y >> 1)) & 0xFFFFFFFF;                    \
  T = ((Y >>  8) ^ X) & 0x00FF00FF; X ^= T; Y ^= (T <<  8);   \
  T = ((Y >>  2) ^ X) & 0x33333333; X ^= T; Y ^= (T <<  2);   \
  T = ((X >> 16) ^ Y) & 0x0000FFFF; Y ^= T; X ^= (T << 16);   \
  T = ((X >>  4) ^ Y) & 0x0F0F0F0F; Y ^= T; X ^= (T <<  4);   \
}

/*
 * DES round macro
 */
#define DES_ROUND(X,Y) {                      \
  T = *SK++ ^ X;                              \
  Y ^= SB8[ (T      ) & 0x3F ] ^              \
       SB6[ (T >>  8) & 0x3F ] ^              \
       SB4[ (T >> 16) & 0x3F ] ^              \
       SB2[ (T >> 24) & 0x3F ];               \
                                              \
  T = *SK++ ^ ((X << 28) | (X >> 4));         \
  Y ^= SB7[ (T      ) & 0x3F ] ^              \
       SB5[ (T >>  8) & 0x3F ] ^              \
       SB3[ (T >> 16) & 0x3F ] ^              \
       SB1[ (T >> 24) & 0x3F ];               \
}

#define SWAP(a,b) { uint32_t t = a; a = b; b = t; t = 0; }

void des_init(des_context *ctx) {
  memset(ctx, 0, sizeof(des_context));
}

void des_free(des_context *ctx) {
  if (ctx == NULL)
    return;

  zeroize(ctx, sizeof(des_context));
}

void des3_init(des3_context *ctx) {
  memset(ctx, 0, sizeof(des3_context));
}

void des3_free(des3_context *ctx) {
  if (ctx == NULL)
    return;

  zeroize(ctx, sizeof(des3_context));
}

static const unsigned char odd_parity_table[128] = { 1,  2,  4,  7,  8,
  11, 13, 14, 16, 19, 21, 22, 25, 26, 28, 31, 32, 35, 37, 38, 41, 42, 44,
  47, 49, 50, 52, 55, 56, 59, 61, 62, 64, 67, 69, 70, 73, 74, 76, 79, 81,
  82, 84, 87, 88, 91, 93, 94, 97, 98, 100, 103, 104, 107, 109, 110, 112,
  115, 117, 118, 121, 122, 124, 127, 128, 131, 133, 134, 137, 138, 140,
  143, 145, 146, 148, 151, 152, 155, 157, 158, 161, 162, 164, 167, 168,
  171, 173, 174, 176, 179, 181, 182, 185, 186, 188, 191, 193, 194, 196,
  199, 200, 203, 205, 206, 208, 211, 213, 214, 217, 218, 220, 223, 224,
  227, 229, 230, 233, 234, 236, 239, 241, 242, 244, 247, 248, 251, 253,
  254 };

void des_key_set_parity(unsigned char key[DES_KEY_SIZE]) {
  int i;

  for (i = 0; i < DES_KEY_SIZE; i++)
    key[i] = odd_parity_table[key[i] / 2];
}

/*
 * Check the given key's parity, returns 1 on failure, 0 on SUCCESS
 */
int des_key_check_key_parity(const unsigned char key[DES_KEY_SIZE]) {
  int i;

  for (i = 0; i < DES_KEY_SIZE; i++)
    if (key[i] != odd_parity_table[key[i] / 2])
      return(1);

  return(0);
}

/*
 * Table of weak and semi-weak keys
 *
 * Source: http://en.wikipedia.org/wiki/Weak_key
 *
 * Weak:
 * Alternating ones + zeros (0x0101010101010101)
 * Alternating 'F' + 'E' (0xFEFEFEFEFEFEFEFE)
 * '0xE0E0E0E0F1F1F1F1'
 * '0x1F1F1F1F0E0E0E0E'
 *
 * Semi-weak:
 * 0x011F011F010E010E and 0x1F011F010E010E01
 * 0x01E001E001F101F1 and 0xE001E001F101F101
 * 0x01FE01FE01FE01FE and 0xFE01FE01FE01FE01
 * 0x1FE01FE00EF10EF1 and 0xE01FE01FF10EF10E
 * 0x1FFE1FFE0EFE0EFE and 0xFE1FFE1FFE0EFE0E
 * 0xE0FEE0FEF1FEF1FE and 0xFEE0FEE0FEF1FEF1
 *
 */

#define WEAK_KEY_COUNT 16

static const unsigned char weak_key_table[WEAK_KEY_COUNT][DES_KEY_SIZE] = {
  { 0x01, 0x01, 0x01, 0x01, 0x01, 0x01, 0x01, 0x01 },
  { 0xFE, 0xFE, 0xFE, 0xFE, 0xFE, 0xFE, 0xFE, 0xFE },
  { 0x1F, 0x1F, 0x1F, 0x1F, 0x0E, 0x0E, 0x0E, 0x0E },
  { 0xE0, 0xE0, 0xE0, 0xE0, 0xF1, 0xF1, 0xF1, 0xF1 },

  { 0x01, 0x1F, 0x01, 0x1F, 0x01, 0x0E, 0x01, 0x0E },
  { 0x1F, 0x01, 0x1F, 0x01, 0x0E, 0x01, 0x0E, 0x01 },
  { 0x01, 0xE0, 0x01, 0xE0, 0x01, 0xF1, 0x01, 0xF1 },
  { 0xE0, 0x01, 0xE0, 0x01, 0xF1, 0x01, 0xF1, 0x01 },
  { 0x01, 0xFE, 0x01, 0xFE, 0x01, 0xFE, 0x01, 0xFE },
  { 0xFE, 0x01, 0xFE, 0x01, 0xFE, 0x01, 0xFE, 0x01 },
  { 0x1F, 0xE0, 0x1F, 0xE0, 0x0E, 0xF1, 0x0E, 0xF1 },
  { 0xE0, 0x1F, 0xE0, 0x1F, 0xF1, 0x0E, 0xF1, 0x0E },
  { 0x1F, 0xFE, 0x1F, 0xFE, 0x0E, 0xFE, 0x0E, 0xFE },
  { 0xFE, 0x1F, 0xFE, 0x1F, 0xFE, 0x0E, 0xFE, 0x0E },
  { 0xE0, 0xFE, 0xE0, 0xFE, 0xF1, 0xFE, 0xF1, 0xFE },
  { 0xFE, 0xE0, 0xFE, 0xE0, 0xFE, 0xF1, 0xFE, 0xF1 }
};

int des_key_check_weak(const unsigned char key[DES_KEY_SIZE]) {
  int i;

  for (i = 0; i < WEAK_KEY_COUNT; i++)
    if (memcmp(weak_key_table[i], key, DES_KEY_SIZE) == 0)
      return(1);

  return(0);
}

static void des_setkey(uint32_t SK[32], const unsigned char key[DES_KEY_SIZE]) {
  int i;
  uint32_t X, Y, T;

  GET_UINT32_BE(X, key, 0);
  GET_UINT32_BE(Y, key, 4);

  /*
   * Permuted Choice 1
   */
  T =   ((Y >>  4) ^ X) & 0x0F0F0F0F;  X ^= T; Y ^= (T <<  4);
  T =   ((Y      ) ^ X) & 0x10101010;  X ^= T; Y ^= (T      );

  X =   (LHs[ (X      ) & 0xF] << 3) | (LHs[ (X >>  8) & 0xF ] << 2)
      | (LHs[ (X >> 16) & 0xF] << 1) | (LHs[ (X >> 24) & 0xF ]     )
      | (LHs[ (X >>  5) & 0xF] << 7) | (LHs[ (X >> 13) & 0xF ] << 6)
      | (LHs[ (X >> 21) & 0xF] << 5) | (LHs[ (X >> 29) & 0xF ] << 4);

  Y =   (RHs[ (Y >>  1) & 0xF] << 3) | (RHs[ (Y >>  9) & 0xF ] << 2)
      | (RHs[ (Y >> 17) & 0xF] << 1) | (RHs[ (Y >> 25) & 0xF ]     )
      | (RHs[ (Y >>  4) & 0xF] << 7) | (RHs[ (Y >> 12) & 0xF ] << 6)
      | (RHs[ (Y >> 20) & 0xF] << 5) | (RHs[ (Y >> 28) & 0xF ] << 4);

  X &= 0x0FFFFFFF;
  Y &= 0x0FFFFFFF;

  /*
   * calculate subkeys
   */
  for (i = 0; i < 16; i++) {
    if (i < 2 || i == 8 || i == 15) {
      X = ((X <<  1) | (X >> 27)) & 0x0FFFFFFF;
      Y = ((Y <<  1) | (Y >> 27)) & 0x0FFFFFFF;
    } else {
      X = ((X <<  2) | (X >> 26)) & 0x0FFFFFFF;
      Y = ((Y <<  2) | (Y >> 26)) & 0x0FFFFFFF;
    }

    *SK++ =   ((X <<  4) & 0x24000000) | ((X << 28) & 0x10000000)
            | ((X << 14) & 0x08000000) | ((X << 18) & 0x02080000)
            | ((X <<  6) & 0x01000000) | ((X <<  9) & 0x00200000)
            | ((X >>  1) & 0x00100000) | ((X << 10) & 0x00040000)
            | ((X <<  2) & 0x00020000) | ((X >> 10) & 0x00010000)
            | ((Y >> 13) & 0x00002000) | ((Y >>  4) & 0x00001000)
            | ((Y <<  6) & 0x00000800) | ((Y >>  1) & 0x00000400)
            | ((Y >> 14) & 0x00000200) | ((Y      ) & 0x00000100)
            | ((Y >>  5) & 0x00000020) | ((Y >> 10) & 0x00000010)
            | ((Y >>  3) & 0x00000008) | ((Y >> 18) & 0x00000004)
            | ((Y >> 26) & 0x00000002) | ((Y >> 24) & 0x00000001);

    *SK++ =   ((X << 15) & 0x20000000) | ((X << 17) & 0x10000000)
            | ((X << 10) & 0x08000000) | ((X << 22) & 0x04000000)
            | ((X >>  2) & 0x02000000) | ((X <<  1) & 0x01000000)
            | ((X << 16) & 0x00200000) | ((X << 11) & 0x00100000)
            | ((X <<  3) & 0x00080000) | ((X >>  6) & 0x00040000)
            | ((X << 15) & 0x00020000) | ((X >>  4) & 0x00010000)
            | ((Y >>  2) & 0x00002000) | ((Y <<  8) & 0x00001000)
            | ((Y >> 14) & 0x00000808) | ((Y >>  9) & 0x00000400)
            | ((Y      ) & 0x00000200) | ((Y <<  7) & 0x00000100)
            | ((Y >>  7) & 0x00000020) | ((Y >>  3) & 0x00000011)
            | ((Y <<  2) & 0x00000004) | ((Y >> 21) & 0x00000002);
  }
}

/*
 * DES key schedule (56-bit, encryption)
 */
int des_setkey_enc(des_context *ctx, const unsigned char key[DES_KEY_SIZE]) {
  des_setkey(ctx->sk, key);

  return(0);
}

/*
 * DES key schedule (56-bit, decryption)
 */
int des_setkey_dec(des_context *ctx, const unsigned char key[DES_KEY_SIZE]) {
  int i;

  des_setkey(ctx->sk, key);

  for (i = 0; i < 16; i += 2) {
    SWAP(ctx->sk[i    ], ctx->sk[30 - i]);
    SWAP(ctx->sk[i + 1], ctx->sk[31 - i]);
  }

  return(0);
}

static void des3_set2key(uint32_t esk[96],
  uint32_t dsk[96],
  const unsigned char key[DES_KEY_SIZE*2]) {

  int i;

  des_setkey(esk, key);
  des_setkey(dsk + 32, key + 8);

  for (i = 0; i < 32; i += 2)
  {
    dsk[i     ] = esk[30 - i];
    dsk[i +  1] = esk[31 - i];

    esk[i + 32] = dsk[62 - i];
    esk[i + 33] = dsk[63 - i];

    esk[i + 64] = esk[i    ];
    esk[i + 65] = esk[i + 1];

    dsk[i + 64] = dsk[i    ];
    dsk[i + 65] = dsk[i + 1];
  }
}

/*
 * Triple-DES key schedule (112-bit, encryption)
 */
int des3_set2key_enc(des3_context *ctx,
  const unsigned char key[DES_KEY_SIZE * 2]) {

  uint32_t sk[96];

  des3_set2key(ctx->sk, sk, key);
  zeroize(sk,  sizeof(sk));

  return(0);
}

/*
 * Triple-DES key schedule (112-bit, decryption)
 */
int des3_set2key_dec(des3_context *ctx,
  const unsigned char key[DES_KEY_SIZE * 2]) {

  uint32_t sk[96];

  des3_set2key(sk, ctx->sk, key);
  zeroize(sk,  sizeof(sk));

  return(0);
}

static void des3_set3key(uint32_t esk[96],
  uint32_t dsk[96],
  const unsigned char key[24]) {

  int i;

  des_setkey(esk, key);
  des_setkey(dsk + 32, key +  8);
  des_setkey(esk + 64, key + 16);

  for (i = 0; i < 32; i += 2) {
    dsk[i     ] = esk[94 - i];
    dsk[i +  1] = esk[95 - i];

    esk[i + 32] = dsk[62 - i];
    esk[i + 33] = dsk[63 - i];

    dsk[i + 64] = esk[30 - i];
    dsk[i + 65] = esk[31 - i];
  }
}

/*
 * Triple-DES key schedule (168-bit, encryption)
 */
int des3_set3key_enc(des3_context *ctx,
  const unsigned char key[DES_KEY_SIZE * 3]) {

  uint32_t sk[96];

  des3_set3key(ctx->sk, sk, key);
  zeroize(sk,  sizeof(sk));

  return(0);
}

/*
 * Triple-DES key schedule (168-bit, decryption)
 */
int des3_set3key_dec(des3_context *ctx,
  const unsigned char key[DES_KEY_SIZE * 3]) {

  uint32_t sk[96];

  des3_set3key(sk, ctx->sk, key);
  zeroize(sk,  sizeof(sk));

  return(0);
}

__constant__ des_context des_ctx;
__constant__ des3_context des3_ctx;

/*
 * DES-ECB block encryption/decryption
 */
__global__ void des_crypt_ecb_kernel(
  const unsigned char *inputs,
  unsigned char *outputs) {

  const unsigned char *input = inputs + TX * DES_BLOCK_SIZE;
  unsigned char *output = outputs + TX * DES_BLOCK_SIZE;

  int i;
  uint32_t X, Y, T, *SK;

  SK = des_ctx.sk;

  GET_UINT32_BE(X, input, 0);
  GET_UINT32_BE(Y, input, 4);

  DES_IP(X, Y);

  for (i = 0; i < 8; i++) {
    DES_ROUND(Y, X);
    DES_ROUND(X, Y);
  }

  DES_FP(Y, X);

  PUT_UINT32_BE(Y, output, 0);
  PUT_UINT32_BE(X, output, 4);
}

int des_transfer_context(des_context *ctx) {
  cuda_upload_symbol(ctx, des_ctx, sizeof(des_context));
  return 0;
}

int des_crypt_ecb(const unsigned char *input,
  size_t length,
  unsigned char *output,
  cuda_device *d) {

  cuda_upload_data(input, d->device_data_in, length);
  
  int grid_size = length / (MAX_THREAD * DES_BLOCK_SIZE);
  if (length % (MAX_THREAD * DES_BLOCK_SIZE) != 0)
    grid_size += 1;
  int thread_size = (length / DES_BLOCK_SIZE) < MAX_THREAD ? 
    length / DES_BLOCK_SIZE : MAX_THREAD;

  // printf("DES_KERNEL<<<%d,%d>>>\n", grid_size, thread_size);

  des_crypt_ecb_kernel<<<grid_size, thread_size>>>(d->device_data_in, d->device_data_out);

  cuda_download_data(output, d->device_data_out, length);

  return 0;
}

/*
 * 3DES-ECB block encryption/decryption
 */
__global__ void des3_crypt_ecb_kernel(
  const unsigned char *inputs,
  unsigned char *outputs) {

  const unsigned char *input = inputs + TX * DES_BLOCK_SIZE;
  unsigned char *output = outputs + TX * DES_BLOCK_SIZE;

  int i;
  uint32_t X, Y, T, *SK;

  SK = des3_ctx.sk;

  GET_UINT32_BE(X, input, 0);
  GET_UINT32_BE(Y, input, 4);

  DES_IP(X, Y);

  for (i = 0; i < 8; i++) {
    DES_ROUND(Y, X);
    DES_ROUND(X, Y);
  }

  for (i = 0; i < 8; i++) {
    DES_ROUND(X, Y);
    DES_ROUND(Y, X);
  }

  for (i = 0; i < 8; i++) {
    DES_ROUND(Y, X);
    DES_ROUND(X, Y);
  }

  DES_FP(Y, X);

  PUT_UINT32_BE(Y, output, 0);
  PUT_UINT32_BE(X, output, 4);
}

int des3_transfer_context(des3_context *ctx) {
  cuda_upload_symbol(ctx, des3_ctx, sizeof(des3_context));
  return 0;
}

int des3_crypt_ecb(
  const unsigned char *input,
  size_t length,
  unsigned char *output,
  cuda_device *d) {

  cuda_upload_data(input, d->device_data_in, length);
  
  int grid_size = length / (MAX_THREAD * DES_BLOCK_SIZE);
  if (length % (MAX_THREAD * DES_BLOCK_SIZE) != 0)
    grid_size += 1;
  int thread_size = (length / DES_BLOCK_SIZE) < MAX_THREAD ? 
    length / DES_BLOCK_SIZE : MAX_THREAD;

  // printf("DES_KERNEL<<<%d,%d>>>\n", grid_size, thread_size);

  des3_crypt_ecb_kernel<<<grid_size, thread_size>>>(d->device_data_in, d->device_data_out);

  cuda_download_data(output, d->device_data_out, length);

  return 0;
}
#if defined(CUDASSL_SELF_TEST)

#include <stdio.h>

/*
 * DES and 3DES test vectors from:
 *
 * http://csrc.nist.gov/groups/STM/cavp/documents/des/tripledes-vectors.zip
 */
static const unsigned char des3_test_keys[24] =
{
  0x01, 0x23, 0x45, 0x67, 0x89, 0xAB, 0xCD, 0xEF,
  0x23, 0x45, 0x67, 0x89, 0xAB, 0xCD, 0xEF, 0x01,
  0x45, 0x67, 0x89, 0xAB, 0xCD, 0xEF, 0x01, 0x23
};

static const unsigned char des3_test_buf[8] =
{
  0x4E, 0x6F, 0x77, 0x20, 0x69, 0x73, 0x20, 0x74
};

static const unsigned char des3_test_ecb_dec[3][8] =
{
  { 0xCD, 0xD6, 0x4F, 0x2F, 0x94, 0x27, 0xC1, 0x5D },
  { 0x69, 0x96, 0xC8, 0xFA, 0x47, 0xA2, 0xAB, 0xEB },
  { 0x83, 0x25, 0x39, 0x76, 0x44, 0x09, 0x1A, 0x0A }
};

static const unsigned char des3_test_ecb_enc[3][8] =
{
  { 0x6A, 0x2A, 0x19, 0xF4, 0x1E, 0xCA, 0x85, 0x4B },
  { 0x03, 0xE6, 0x9F, 0x5B, 0xFA, 0x58, 0xEB, 0x42 },
  { 0xDD, 0x17, 0xE8, 0xB8, 0xB4, 0x37, 0xD2, 0x32 }
};

/*
 * Checkup routine
 */
extern "C" int des_self_test(int verbose, cuda_device *d)
{
  int i, j, u, v, ret = 0;
  des_context ctx;
  des3_context ctx3;
  unsigned char buf[MAX_THREAD][DES_BLOCK_SIZE];

  des_init(&ctx);
  des3_init(&ctx3);

  /*
   * ECB mode
   */
  for (i = 0; i < 6; i++) {
    u = i >> 1;
    v = i  & 1;

    if (verbose != 0)
      printf("  DES%c-ECB-%3d (%s): ",
       (u == 0) ? ' ' : '3', 56 + u * 56,
       (v == DES_DECRYPT) ? "dec" : "enc");

    memcpy(buf[0], des3_test_buf, DES_BLOCK_SIZE);
    memcpy(buf[1], des3_test_buf, DES_BLOCK_SIZE);

    switch (i) {
    case 0:
      des_setkey_dec(&ctx, des3_test_keys);
      des_transfer_context(&ctx);
      break;

    case 1:
      des_setkey_enc(&ctx, des3_test_keys);
      des_transfer_context(&ctx);
      break;

    case 2:
      des3_set2key_dec(&ctx3, des3_test_keys);
      des3_transfer_context(&ctx3);
      break;

    case 3:
      des3_set2key_enc(&ctx3, des3_test_keys);
      des3_transfer_context(&ctx3);
      break;

    case 4:
      des3_set3key_dec(&ctx3, des3_test_keys);
      des3_transfer_context(&ctx3);
      break;

    case 5:
      des3_set3key_enc(&ctx3, des3_test_keys);
      des3_transfer_context(&ctx3);
      break;

    default:
      return(1);
    }

    for (j = 0; j < 10000; j++) {
      if (u == 0)
        des_crypt_ecb(*buf, DES_BLOCK_SIZE * 2, *buf, d);
      else
        des3_crypt_ecb(*buf, DES_BLOCK_SIZE * 2, *buf, d);
    }

    if (v == DES_DECRYPT) {
      if (memcmp(buf[0], des3_test_ecb_dec[u], DES_BLOCK_SIZE) != 0 && 
        memcmp(buf[1], des3_test_ecb_dec[u], DES_BLOCK_SIZE) != 0) {
        if (verbose != 0)
          printf("failed\n");

        ret = 1;
        goto exit;
      }
    } else {
      if (memcmp(buf[0], des3_test_ecb_enc[u], DES_BLOCK_SIZE) != 0 &&
        memcmp(buf[1], des3_test_ecb_enc[u], DES_BLOCK_SIZE) != 0) {
        if (verbose != 0)
          printf("failed\n");

        ret = 1;
        goto exit;
      }
    }

    if (verbose != 0)
      printf("passed\n");
  }

  if (verbose != 0)
    printf("\n");

exit:
  des_free(&ctx);
  des3_free(&ctx3);

  return(ret);
}

#define DATASIZE 1000L
#define LOOPS 1000L
extern "C" int des_performance_test(int verbose, cuda_device *d) {
  unsigned char key[DES_KEY_SIZE];
  unsigned char buf[MAX_THREAD * DATASIZE][DES_BLOCK_SIZE];
  des_context ctx;
  int i; float h;

  CUDA_START_TIME

  memset(key, 0, DES_KEY_SIZE);
  memset(buf, 0, MAX_THREAD * DES_BLOCK_SIZE * DATASIZE);
  des_init(&ctx);

  des_setkey_enc(&ctx, key);
  des_transfer_context(&ctx);

  for (int i = 0; i < LOOPS; ++i)
    des_crypt_ecb(*buf, MAX_THREAD * DES_BLOCK_SIZE * DATASIZE, *buf, d);

  CUDA_STOP_TIME("  DES -ECB- 56 (enc)")
  printf("    Block Data size: %ld\n", MAX_THREAD * DES_BLOCK_SIZE * DATASIZE);
  printf("    Block Loops: %ld\n", LOOPS);

  TALK_LIKE_A_HUMAN_BEING(MAX_THREAD * DES_BLOCK_SIZE * DATASIZE * LOOPS, "    ", " in total\n");
  TALK_LIKE_A_HUMAN_BEING(MAX_THREAD * DES_BLOCK_SIZE * DATASIZE * LOOPS / gpu_time * 1000, "    ", "/sec\n");
  
  printf("    %ld loops in total\n", LOOPS * MAX_THREAD * DATASIZE);
  printf("    %f loops/sec\n", LOOPS * MAX_THREAD * DATASIZE / gpu_time * 1000);

  if (verbose != 0)
    printf("\n");

  return 0;
}

extern "C" int des3_performance_test(int verbose, cuda_device *d) {
  unsigned char key[DES_KEY_SIZE];
  unsigned char buf[MAX_THREAD * DATASIZE][DES_BLOCK_SIZE];
  des3_context ctx;
  int i; float h;

  CUDA_START_TIME

  memset(key, 0, DES_KEY_SIZE);
  memset(buf, 0, MAX_THREAD * DES_BLOCK_SIZE * DATASIZE);
  des3_init(&ctx);

  des3_set3key_enc(&ctx, key);
  des3_transfer_context(&ctx);

  for (int i = 0; i < LOOPS; ++i)
    des3_crypt_ecb(*buf, MAX_THREAD * DES_BLOCK_SIZE * DATASIZE, *buf, d);

  CUDA_STOP_TIME("  DES3-ECB-168 (enc)")
  printf("    Block Data size: %ld\n", MAX_THREAD * DES_BLOCK_SIZE * DATASIZE);
  printf("    Block Loops: %ld\n", LOOPS);

  TALK_LIKE_A_HUMAN_BEING(MAX_THREAD * DES_BLOCK_SIZE * DATASIZE * LOOPS, "    ", " in total\n");
  TALK_LIKE_A_HUMAN_BEING(MAX_THREAD * DES_BLOCK_SIZE * DATASIZE * LOOPS / gpu_time * 1000, "    ", "/sec\n");
  
  printf("    %ld loops in total\n", LOOPS * MAX_THREAD * DATASIZE);
  printf("    %f loops/sec\n", LOOPS * MAX_THREAD * DATASIZE / gpu_time * 1000);

  if (verbose != 0)
    printf("\n");

  return 0;
}
#endif /* CUDASSL_SELF_TEST */

#endif /* CUDASSL_DES_C */
